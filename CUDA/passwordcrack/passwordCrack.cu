#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>


__device__ int is_a_match(char *attempt) {
	char mypassword1[] = "AV4567";
	char mypassword2[] = "FG7868";
	char mypassword3[] = "HJ7654";
	char mypassword4[] = "DE6789";


	char *b = attempt;
	char *i = attempt;
	char *s = attempt;
	char *h = attempt;
	char *p1 = mypassword1;
	char *p2 = mypassword2;
	char *p3 = mypassword3;
	char *p4 = mypassword4;

	while(*b == *p1) { 
		if(*b == '\0') 
		{
			printf("Password: %s\n",mypassword1);
			break;
		}

		b++;
		p1++;
	}
	
	while(*i == *p2) { 
		if(*i == '\0') 
		{
			printf("Password: %s\n",mypassword2);
			break;
		}

		i++;
		p2++;
	}

	while(*s == *p3) { 
		if(*s == '\0') 
		{
			printf("Password: %s\n",mypassword3);
			break;
		}

		s++;
		p3++;
	}

	while(*h == *p4) { 
		if(*h == '\0') 
		{
			printf("Password: %s\n",mypassword4);
			return 1;
		}

		h++;
		p4++;
	}
	return 0;

}

__global__ void  kernel() {
	char i1,i2,i3,i4;

	char password[7];
	password[6] = '\0';

	int i = blockIdx.x+65;
	int j = threadIdx.x+65;
	char firstMatch = i; 
	char secondMatch = j; 

	password[0] = firstMatch;
	password[1] = secondMatch;
	for(i1='0'; i1<='9'; i1++){
		for(i2='0'; i2<='9'; i2++){
			for(i3='0'; i3<='9'; i3++){
				for(i4='0'; i4<='9'; i4++){
					password[2] = i1;
					password[3] = i2;
					password[4] = i3;
					password[5] = i4; 
					if(is_a_match(password)) {
					} 
					else {
	     			//printf("tried: %s\n", password);		  
					}
				}
			}
		}
	}
}

int time_difference(struct timespec *start, 
	struct timespec *finish, 
	long long int *difference) {
	long long int ds =  finish->tv_sec - start->tv_sec; 
	long long int dn =  finish->tv_nsec - start->tv_nsec; 
	if(dn < 0 ) {
		ds--;
		dn += 1000000000; 
	} 
	*difference = ds * 1000000000 + dn;
	return !(*difference > 0);
}


int main() {

	struct  timespec start, finish;
	long long int time_elapsed;
	clock_gettime(CLOCK_MONOTONIC, &start);

	kernel <<<26,26>>>();
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

	return 0;
}


